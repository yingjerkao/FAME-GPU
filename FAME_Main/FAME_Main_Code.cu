#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_FFT_CUDA.cuh"
#include "FAME_Create_cublas.cuh"
#include "FAME_Create_Buffer_Isotropic.cuh"
#include "FAME_Create_Buffer_Anisotropic.cuh"
#include "FAME_Create_Buffer_Biisotropic.cuh"
#include "FAME_Create_Buffer_Bianisotropic.cuh"
#include "FAME_Matrix_B_Isotropic.cuh"
#include "FAME_Matrix_B_Anisotropic.cuh"
#include "FAME_Matrix_B_Biisotropic.cuh"
#include "FAME_Matrix_B_Bianisotropic.cuh"
#include "FAME_Malloc_mtx_C.h"
#include "FAME_Matrix_Lambdas.cuh"
#include "FAME_Matrix_Curl.h"
#include "FAME_Create_Frequency_txt.h"
#include "FAME_Save_Eigenvector.h"
#include "FAME_Profile.h"
#include "FAME_Destroy_Main.cuh"
#include "FAME_Fast_Algorithms_Isotropic.cuh"
#include "FAME_Fast_Algorithms_Anisotropic.cuh"
#include "FAME_Fast_Algorithms_Biisotropic.cuh"
#include "FAME_Fast_Algorithms_Bianisotropic.cuh"
#include "FAME_Matrix_Vector_Production_Qrs.cuh"
#include "FAME_Matrix_Vector_Production_Pr.cuh"
#include "FAME_Print_Parameter.h"
#include "FAME_Create_Lambdas_txt.h"
#include "vec_plus.h"
#include "vec_norm.h"
#include "vec_inner_prod.h"
#include "mtx_print.h"
#include "mtx_prod.h"
#include "mtx_trans.h"
#include "mtx_trans_conj.h"
#include "mtx_cat.h"
#include "mtx_dot_prod.h"
#include "kron_vec.h"
#include "inv3.h"
#include "printDeviceArray.cuh"

void FAME_Fast_Algorithms_Driver(
	realCPU*        Freq_array,
	cmpxCPU*        Ele_field_mtx,
	cmpxCPU*        Dis_field_mtx,
	CULIB_HANDLES   cuHandles,
	LANCZOS_BUFFER  lBuffer,
	FFT_BUFFER      fft_buffer,
	LAMBDAS_CUDA    Lambdas_cuda,
	MTX_B           mtx_B,
	MATERIAL 	    material,
	ES              es,
	LS              ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* lattice_type, PROFILE* Profile);

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type);
void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, realCPU error, PROFILE* Profile);
void Check_Residual_Iso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error);
void Check_Residual_Aniso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, cmpxCPU* Dis_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error);
void Check_Residual_Biiso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error);
void Check_Residual_Bianiso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, cmpxCPU* Dis_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error);

int FAME_Main_Code(PAR Par, PROFILE* Profile)
{

	int Nx = Par.mesh.grid_nums[0];
    int Ny = Par.mesh.grid_nums[1];
	int Nz = Par.mesh.grid_nums[2];
	int Nd;
	int N  = Nx * Ny * Nz;
	int Ele_field_mtx_N = 0;
	int Nwant = Par.es.nwant;
	int Nstep = Par.es.nstep;
	int N_wave_vec = Par.recip_lattice.Wave_vec_num;
	realCPU wave_vec_array[3];
	
	#if defined(USE_SINGLE)
		Par.ce_error = 1e-1;
		Par.es.tol = 1e-6;
		Par.ls.tol = 1e-6;
		realCPU cr_error = 1e-3;
	#else
		Par.ce_error = 1e-7;
		Par.es.tol = 1e-12;
		Par.ls.tol = 1e-12;
		realCPU cr_error = 1e-8;
	#endif 

	struct timespec start, end;
	realCPU accum;

	hipSetDevice(Par.flag.device);
	
    CULIB_HANDLES  cuHandles;
	FFT_BUFFER     fft_buffer;
	LANCZOS_BUFFER lBuffer;
	MTX_B          mtx_B;
	MTX_C          mtx_C;
	LAMBDAS        Lambdas;
    LAMBDAS_CUDA   Lambdas_cuda;

	FAME_Create_cublas(&cuHandles, Nx, Ny, Nz);

	realCPU *Freq_array;
	cmpxCPU *Ele_field_mtx, *Dis_field_mtx;
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		Ele_field_mtx_N = N * 3;
		Freq_array = (realCPU*) calloc(N_wave_vec * Nwant, sizeof(realCPU));
		Ele_field_mtx = (cmpxCPU*) calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
	}
	else if(strcmp(Par.material.material_type, "anisotropic") == 0)
	{
		Ele_field_mtx_N = N * 12;
		Freq_array = (realCPU*) calloc(N_wave_vec * Nwant, sizeof(realCPU));
		Ele_field_mtx = (cmpxCPU*) calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
		Dis_field_mtx = (cmpxCPU*) calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
	}
	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
	{
		Ele_field_mtx_N = N * 6;
		Freq_array = (realCPU*) calloc(N_wave_vec * Nwant, sizeof(realCPU));
		Ele_field_mtx = (cmpxCPU*) calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
	}
	else if(strcmp(Par.material.material_type, "bianisotropic") == 0)
	{
		Ele_field_mtx_N = N * 48;
		Freq_array = (realCPU*) calloc(N_wave_vec * Nwant, sizeof(realCPU));
		Ele_field_mtx = (cmpxCPU*) calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
		Dis_field_mtx = (cmpxCPU*) calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
	}
	
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		FAME_Create_Buffer_Isotropic(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	}
	else if(strcmp(Par.material.material_type, "anisotropic") == 0)
	{
		FAME_Create_Buffer_Anisotropic(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	}
	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
	{
		FAME_Create_Buffer_Biisotropic(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	}
	else if(strcmp(Par.material.material_type, "bianisotropic") == 0)
	{
		FAME_Create_Buffer_Bianisotropic(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	}

	
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		printf("= = = = FAME_Matrix_B_Isotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		size_t size = 3*N*sizeof(realCPU);
		checkCudaErrors(hipMalloc((void**) &mtx_B.B_eps,    size));
		checkCudaErrors(hipMalloc((void**) &mtx_B.invB_eps, size));
		FAME_Matrix_B_Isotropic(mtx_B.B_eps, mtx_B.invB_eps, Par.material, N);
	}

	else if(strcmp(Par.material.material_type, "anisotropic") == 0)
    {
		printf("= = = = FAME_Matrix_B_Anisotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		hipMalloc((void**) &mtx_B.N, 9 * sizeof(cmpxGPU));
		hipMalloc((void**) &mtx_B.GInOut_index,  Par.material.BInOut_index_length[7] * sizeof(int));
		mtx_B.GInOut_index_length = (int *) calloc(8, sizeof(int));
    	hipMemcpy(mtx_B.GInOut_index, Par.material.BInOut_index, Par.material.BInOut_index_length[7] * sizeof(int), hipMemcpyHostToDevice);
    	memcpy(mtx_B.GInOut_index_length, Par.material.BInOut_index_length, 8 * sizeof(int));

		FAME_Matrix_B_Anisotropic(N, Par.material, mtx_B.N );
	}

	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
    {
		printf("= = = = FAME_Matrix_B_Biisotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		size_t size = 3 * N * sizeof(realGPU);
		hipMalloc((void**) &mtx_B.B_eps, size);
		hipMalloc((void**) &mtx_B.B_mu, size);
		hipMalloc((void**) &mtx_B.invPhi, size);

		size = 3 * N * sizeof(cmpxGPU);
		hipMalloc((void**) &mtx_B.B_zeta, size);
		hipMalloc((void**) &mtx_B.B_zeta_s, size);
		hipMalloc((void**) &mtx_B.B_xi, size);

		FAME_Matrix_B_Biisotropic(N, Par.material, mtx_B.B_eps, mtx_B.B_mu, mtx_B.B_xi, mtx_B.B_zeta, mtx_B.B_zeta_s, mtx_B.invPhi);
	}

	else if(strcmp(Par.material.material_type, "bianisotropic") == 0)
    {
		printf("= = = = FAME_Matrix_B_Bianisotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		hipMalloc((void**) &mtx_B.G, 36 * sizeof(cmpxGPU));
		hipMalloc((void**) &mtx_B.GInOut_index,  Par.material.BInOut_index_length[7] * sizeof(int));
		mtx_B.GInOut_index_length = (int *) calloc(8, sizeof(int));
    	hipMemcpy(mtx_B.GInOut_index, Par.material.BInOut_index, Par.material.BInOut_index_length[7] * sizeof(int), hipMemcpyHostToDevice);
    	memcpy(mtx_B.GInOut_index_length, Par.material.BInOut_index_length, 8 * sizeof(int));

		FAME_Matrix_B_Bianisotropic(N, Par.material, mtx_B.G);
	}


    FAME_Malloc_mtx_C(&mtx_C, N);
	FAME_Print_Parameter(Par);
	for(int i = 0; i < N_wave_vec; i++)
	{
		Profile->idx = i;

	    wave_vec_array[0] = Par.recip_lattice.WaveVector[3 * i];
    	wave_vec_array[1] = Par.recip_lattice.WaveVector[3 * i + 1];
    	wave_vec_array[2] = Par.recip_lattice.WaveVector[3 * i + 2];

    	printf("\033[40;33m= = Start to compute (%3d/%3d) WaveVector = [ % .6f % .6f % .6f ] = =\033[0m\n", i + 1, Par.recip_lattice.Wave_vec_num, wave_vec_array[0], wave_vec_array[1], wave_vec_array[2]);

		printf("= = = = FAME_Matrix_Curl  = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		FAME_Matrix_Curl(&mtx_C, wave_vec_array, Par.mesh.grid_nums, Par.mesh.edge_len, Par.mesh.mesh_len, Par.lattice);

		printf("= = = = FAME_Matrix_Lambdas = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		Nd = FAME_Matrix_Lambdas(&Lambdas_cuda, wave_vec_array, Par.mesh.grid_nums, Par.mesh.mesh_len, Par.lattice.lattice_vec_a, &Par, &Lambdas);
		
		printf("= = = = Check_Eigendecomp = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime(CLOCK_REALTIME, &start);
		Check_Eigendecomp(mtx_C, Lambdas, Lambdas_cuda, fft_buffer, cuHandles, Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Par.ce_error, Profile);
		clock_gettime(CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		printf("= = = = FAME_Fast_Algorithms = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		FAME_Fast_Algorithms_Driver(Freq_array + i * Nwant, Ele_field_mtx, Dis_field_mtx, 
			cuHandles, lBuffer, fft_buffer, Lambdas_cuda, mtx_B, Par.material, Par.es, Par.ls,
			Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Profile);
		clock_gettime (CLOCK_REALTIME, &end);
		Profile->es_time[Profile->idx] = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;
		
		printf("= = = = Check_Residual  = = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		if(strcmp(Par.material.material_type, "isotropic") == 0)
		{
			Check_Residual_Iso(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant, cr_error);
		}
		else if(strcmp(Par.material.material_type, "anisotropic") == 0)
		{
			Check_Residual_Aniso(Freq_array+i*Nwant, Ele_field_mtx, Dis_field_mtx, mtx_B, mtx_C, N, Nwant, cr_error);
		}
		else if(strcmp(Par.material.material_type, "biisotropic") == 0)
		{
			Check_Residual_Biiso(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant, cr_error);
		}
		else if(strcmp(Par.material.material_type, "bianisotropic") == 0)
		{
			Check_Residual_Bianiso(Freq_array+i*Nwant, Ele_field_mtx, Dis_field_mtx, mtx_B, mtx_C, N, Nwant, cr_error);
		}
		clock_gettime (CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);
		

		if(Par.flag.save_eigen_vector)
		{
			printf("= = = = Save Eigen Vector = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
			FAME_Save_Eigenvector(Ele_field_mtx, Nwant, Ele_field_mtx_N, i);
		}

		Destroy_Lambdas(Lambdas, Lambdas_cuda, Par.lattice.lattice_type);

		FAME_Print_Profile(*Profile);

	}

	FAME_Create_Frequency_txt(Freq_array, Nwant, Profile->idx);
	
	FAME_Destroy_Main(cuHandles, fft_buffer, lBuffer, mtx_B, mtx_C, Freq_array, Ele_field_mtx);
	
	return 0;
}

void FAME_Fast_Algorithms_Driver(
	realCPU*        Freq_array,
	cmpxCPU*        Ele_field_mtx,
	cmpxCPU*        Dis_field_mtx, 
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	MATERIAL 	   material,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* lattice_type, PROFILE* Profile)
{

	if(strcmp(material.material_type, "isotropic") == 0)
	{
		if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
		{

			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								    Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "Simple", Profile);
		}
		else
		{
			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								 	Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "General", Profile);
		}
	}
	else if (strcmp(material.material_type, "anisotropic") == 0)
	{
		if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
			(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
			(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
		{		
			FAME_Fast_Algorithms_Anisotropic(Freq_array, Ele_field_mtx, Dis_field_mtx, cuHandles, lBuffer, fft_buffer,
									Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "Simple", Profile);		
		}
		else
		{
			FAME_Fast_Algorithms_Anisotropic(Freq_array, Ele_field_mtx, Dis_field_mtx, cuHandles, lBuffer, fft_buffer,
									Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "General", Profile);			
		}
	}
	else if (strcmp(material.material_type, "biisotropic") == 0)
	{
		if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
			(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
			(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
		{		
			FAME_Fast_Algorithms_Biisotropic(Freq_array, Ele_field_mtx, cuHandles, Lambdas_cuda, lBuffer, fft_buffer, 
									mtx_B, material, Nx, Ny, Nz, Nd, es, ls, "Simple", Profile);		
		}
		else
		{
			FAME_Fast_Algorithms_Biisotropic(Freq_array, Ele_field_mtx, cuHandles,Lambdas_cuda, lBuffer, fft_buffer, 
									mtx_B, material, Nx, Ny, Nz, Nd, es, ls, "General", Profile);			
		}
	}
	else if (strcmp(material.material_type, "bianisotropic") == 0)
	{
		if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
			(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
			(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
		{		
			FAME_Fast_Algorithms_Bianisotropic(Freq_array, Ele_field_mtx, Dis_field_mtx, cuHandles, Lambdas_cuda, lBuffer, fft_buffer, 
									mtx_B, material ,Nx, Ny, Nz, Nd, es, ls, "Simple", Profile);		
		}
		else
		{
			FAME_Fast_Algorithms_Bianisotropic(Freq_array, Ele_field_mtx, Dis_field_mtx, cuHandles, Lambdas_cuda, lBuffer, fft_buffer, 
									mtx_B, material ,Nx, Ny, Nz, Nd, es, ls , "General", Profile);			
		}
	}
}

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type)
{
	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
        free(Lambdas.D_k);
        free(Lambdas.D_ks);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_k);
		hipFree(Lambdas_cuda.dD_ks);
	}

	else
	{
        free(Lambdas.D_kx);
        free(Lambdas.D_ky);
        free(Lambdas.D_kz);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_kx);
    	hipFree(Lambdas_cuda.dD_ky);
    	hipFree(Lambdas_cuda.dD_kz);
	}

    free(Lambdas.Lambda_q_sqrt);
    free(Lambdas.Pi_Qr);
    free(Lambdas.Pi_Pr);
    free(Lambdas.Pi_Qrs);
    free(Lambdas.Pi_Prs);
	free(Lambdas.Pi_Qr_110);
    free(Lambdas.Pi_Pr_110);
    free(Lambdas.Pi_Qrs_110);
    free(Lambdas.Pi_Prs_110);
	free(Lambdas.Pi_Qr_101);
    free(Lambdas.Pi_Pr_101);
    free(Lambdas.Pi_Qrs_101);
    free(Lambdas.Pi_Prs_101);
	free(Lambdas.Pi_Qr_011);
    free(Lambdas.Pi_Pr_011);
    free(Lambdas.Pi_Qrs_011);
    free(Lambdas.Pi_Prs_011);

    hipFree(Lambdas_cuda.Lambda_q_sqrt);
	hipFree(Lambdas_cuda.dPi_Qr);
	hipFree(Lambdas_cuda.dPi_Pr);
	hipFree(Lambdas_cuda.dPi_Qrs);
	hipFree(Lambdas_cuda.dPi_Prs);
	hipFree(Lambdas_cuda.dPi_Qr_110);
	hipFree(Lambdas_cuda.dPi_Pr_110);
	hipFree(Lambdas_cuda.dPi_Qrs_110);
	hipFree(Lambdas_cuda.dPi_Prs_110);
	hipFree(Lambdas_cuda.dPi_Qr_101);
	hipFree(Lambdas_cuda.dPi_Pr_101);
	hipFree(Lambdas_cuda.dPi_Qrs_101);
	hipFree(Lambdas_cuda.dPi_Prs_101);
	hipFree(Lambdas_cuda.dPi_Qr_011);
	hipFree(Lambdas_cuda.dPi_Pr_011);
	hipFree(Lambdas_cuda.dPi_Qrs_011);
	hipFree(Lambdas_cuda.dPi_Prs_011);
}

void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, realCPU error, PROFILE* Profile)
{
	int i;
	int N2 = N * 2;
	int Ele_field_mtx_N = N * 3;
	int N12 = N * 12;
	size_t size, dsizeEle_field_mtx_N, dsizeNd2;

	size = Ele_field_mtx_N * sizeof(cmpxCPU);

	cmpxCPU* vec_x    = (cmpxCPU*) malloc(size);
	cmpxCPU* vec_y    = (cmpxCPU*) malloc(size);
	cmpxCPU* vec_temp = (cmpxCPU*) malloc(size);

	cmpxGPU* N3_temp1 = cuHandles.N3_temp1;
	cmpxGPU* N3_temp2 = cuHandles.N3_temp2;

	cmpxGPU* Nd2_temp;
	dsizeEle_field_mtx_N = Ele_field_mtx_N * sizeof(cmpxGPU);
	dsizeNd2 = Nd * 2 * sizeof(cmpxGPU);

	checkCudaErrors(hipMalloc((void**)&Nd2_temp, dsizeNd2));

	srand(time(NULL));

	for(i = 0; i < Ele_field_mtx_N; i++)
	//vec_x[i] = ((realCPU) rand()/(RAND_MAX + 1.0))  for test
		vec_x[i] = ((realCPU) rand()/(RAND_MAX + 1.0)) +  I * ((realCPU) rand()/(RAND_MAX + 1.0));

  realCPU vec_x_norm = vec_norm(vec_x, Ele_field_mtx_N);
  /*for(i = 0; i < Ele_field_mtx_N; i++)
    vec_x[i] = vec_x[i] /vec_x_norm;*/
  
	cmpxCPU *vec_y_1, *vec_y_2, *vec_y_3;

	hipMemcpy(N3_temp1, vec_x, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);
  
  //printDeviceArray(N3_temp1, Ele_field_mtx_N, "vec_x.txt");

	if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
	{
		//FFT_CUDA(cuHandles, fft_buffer, N3_temp1, Nx, Ny, Nz, Lambdas_cuda.dD_ks, N3_temp2);
		FFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_ks, fft_buffer, cuHandles, Nx, Ny, Nz);
	}
	else
	{
		for(i = 0; i < 3; i++)
        	spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, -1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);
	vec_y_1 = &vec_y[0];  vec_y_2 = &vec_y[N];  vec_y_3 = &vec_y[N2];

	if(Nd == N - 1)
	{
		vec_y_1[0] = 0; vec_y_2[0] = 0; vec_y_3[0] = 0;
		for(i = 0; i < N - 1; i++)
		{
			vec_y_1[i + 1] = Lambdas.Lambda_x[i] * vec_y_1[i + 1];
			vec_y_2[i + 1] = Lambdas.Lambda_y[i] * vec_y_2[i + 1];
			vec_y_3[i + 1] = Lambdas.Lambda_z[i] * vec_y_3[i + 1];
		}
	}
	else
	{
		for(i = 0; i < N; i++)
		{
			vec_y_1[i] = Lambdas.Lambda_x[i] * vec_y_1[i];
			vec_y_2[i] = Lambdas.Lambda_y[i] * vec_y_2[i];
			vec_y_3[i] = Lambdas.Lambda_z[i] * vec_y_3[i];
		}
	}

	hipMemcpy(N3_temp1, vec_y, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		IFFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_k, fft_buffer, cuHandles, Nx, Ny, Nz);
		//IFFT_CUDA(cuHandles, fft_buffer, N3_temp1, Nx, Ny, Nz, Lambdas_cuda.dD_k, N3_temp2);
	}
	else
	{
		for(i = 0; i < 3; i++)
			spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, 1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);

	mtx_prod(&vec_temp[0] , mtx_C.C1_r, mtx_C.C1_c, mtx_C.C1_v, &vec_x[0] , N2, N);
	mtx_prod(&vec_temp[N] , mtx_C.C2_r, mtx_C.C2_c, mtx_C.C2_v, &vec_x[N] , N2, N);
	mtx_prod(&vec_temp[N2], mtx_C.C3_r, mtx_C.C3_c, mtx_C.C3_v, &vec_x[N2], N2, N);

	size = N * sizeof(cmpxCPU);
	cmpxCPU* test_x = (cmpxCPU*) malloc(size);
	cmpxCPU* test_y = (cmpxCPU*) malloc(size);
	cmpxCPU* test_z = (cmpxCPU*) malloc(size);

	vec_plus(test_x, 1.0, &vec_temp[0] , -1.0, &vec_y[0] , N);
	vec_plus(test_y, 1.0, &vec_temp[N] , -1.0, &vec_y[N] , N);
	vec_plus(test_z, 1.0, &vec_temp[N2], -1.0, &vec_y[N2], N);
	
	realCPU C1_error = vec_norm(test_x, N)/sqrt(N);
    realCPU C2_error = vec_norm(test_y, N)/sqrt(N);
    realCPU C3_error = vec_norm(test_z, N)/sqrt(N);

	free(test_x); free(test_y); free(test_z);

	cmpxCPU* Qrs_x = (cmpxCPU*) malloc(2*Nd*sizeof(cmpxCPU));

	hipMemcpy(N3_temp1, vec_x, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_ks, Lambdas_cuda.dPi_Qrs);
	}
	else
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Qrs);
	}
  
//  printDeviceArray(Nd2_temp, 2*Nd, "QRS.txt");

	hipMemcpy(Qrs_x, Nd2_temp, dsizeNd2, hipMemcpyDeviceToHost);

	for(i = 0; i < Nd; i++ )
	{
		Qrs_x[i]      = Qrs_x[i]      * Lambdas.Lambda_q_sqrt[i];
		Qrs_x[i + Nd] = Qrs_x[i + Nd] * Lambdas.Lambda_q_sqrt[i];
	}

	hipMemcpy(Nd2_temp, Qrs_x, dsizeNd2, hipMemcpyHostToDevice);


	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Pr(N3_temp1, Nd2_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_k, Lambdas_cuda.dPi_Pr);
	}
	else
	{
		FAME_Matrix_Vector_Production_Pr(N3_temp1, Nd2_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Pr);
	}
  
//  printDeviceArray(N3_temp1, Ele_field_mtx_N, "PrS.txt");
//getchar();
  
	hipMemcpy(vec_y, N3_temp1, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);

	mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_x, N12, Ele_field_mtx_N);
	
/*cout<<"vec_temp "<<creal(vec_temp[0] )<<"  "<<cimag(vec_temp[0] )<<endl;
cout<<"vec_temp "<<creal(vec_temp[1] )<<"  "<<cimag(vec_temp[1] )<<endl;
cout<<" vec_x "<<creal( vec_x[0] )<<"  "<<cimag( vec_x[0] )<<endl;
cout<<" vec_x "<<creal( vec_x[1] )<<"  "<<cimag( vec_x[1] )<<endl;*/

	cmpxCPU* test = (cmpxCPU*) malloc(Ele_field_mtx_N * sizeof(cmpxCPU));
	vec_plus(test, 1.0, vec_temp, -1.0, vec_y, Ele_field_mtx_N);
	realCPU SVD_test_C = vec_norm(test, Ele_field_mtx_N)/sqrt(Ele_field_mtx_N);

	printf("          EigDecomp_test_C1 = %e\n", C1_error);
    printf("          EigDecomp_test_C2 = %e\n", C2_error);
    printf("          EigDecomp_test_C3 = %e\n", C3_error);
	printf("          SVD_test_C        = %e\n", SVD_test_C);

  //printDeviceArray(N3_temp1, Ele_field_mtx_N, "vec_y.txt");
//getchar();
	if(C1_error > error || C2_error > error || C3_error > error || SVD_test_C > error)
	{
		printf("\033[40;31mFAME_Main_Code(330):\033[0m\n");
		printf("\033[40;31mThe eigen decomposition is not correct.\033[0m\n");
		printf("\033[40;31mIf N = Nx * Ny * Nz > 256^3, may be caused by numerical errors, please loosen 1e-6.\n");
		printf("\033[40;31mIf not, please contact us.\033[0m\n");
		assert(0);
	}
	
  
	hipFree(Nd2_temp);
	free(test); free(vec_temp); free(Qrs_x);
	free(vec_x); free(vec_y);
}

void Check_Residual_Iso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error)
{
	int Ele_field_mtx_N = N * 3;
	int N12 = N * 12;
	size_t size;

	size = Ele_field_mtx_N * Nwant * sizeof(cmpxCPU);

	cmpxCPU* vec_temp = (cmpxCPU*)malloc(size);
	cmpxCPU* vec_left = (cmpxCPU*)malloc(size);
	cmpxCPU* residual = (cmpxCPU*)malloc(size);

	realCPU res, omega2, res_inf;
	realCPU* B_eps = (realCPU*)calloc(Ele_field_mtx_N, sizeof(realCPU));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, Ele_field_mtx_N*sizeof(realCPU), hipMemcpyDeviceToHost));

	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -pow(Freq_array[i], 2);
		mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*Ele_field_mtx_N, N12, Ele_field_mtx_N);
		mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_temp, N12, Ele_field_mtx_N, "Conjugate Transpose");
		mtx_dot_prod(B_eps, Ele_field_mtx + i*Ele_field_mtx_N, vec_temp, Ele_field_mtx_N, 1);
   
   		vec_plus(residual, 1.0, vec_left, omega2, vec_temp, Ele_field_mtx_N);
		res = vec_norm(residual, Ele_field_mtx_N);
 
   		res_inf = cabs(residual[0]);
   		for (int j = 1; j < Ele_field_mtx_N; j++)
   		{
       		if(cabs(residual[j]) > res_inf)
        	res_inf = cabs(residual[j]);
    	}
    
		if(res > error)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e\033[0m ,residual_inf = %e.\033[0m\n", i, Freq_array[i], res, res_inf);
			// Freq_array[i] = -Freq_array[i];
		}
		else
			printf("Freq(%2d) = %10.8f, residual = %e, residual_inf = %e.\033[0m\n", i, Freq_array[i], res, res_inf);
	}	

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);
}

void Check_Residual_Aniso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, cmpxCPU* Dis_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error)
{
	int Ele_field_mtx_N = N * 12;
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size;

	size = Ele_field_mtx_N * sizeof(cmpxCPU);

	cmpxCPU* vec_temp = (cmpxCPU*)malloc(size);
	cmpxCPU* vec_left = (cmpxCPU*)malloc(size);
	cmpxCPU* residual = (cmpxCPU*)malloc(size);

	realCPU res, omega2;

	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -pow(Freq_array[i], 2);
		mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*Ele_field_mtx_N, N12, N3);
		mtx_prod(vec_temp + N3, mtx_C.C_110_r, mtx_C.C_110_c, mtx_C.C_110_v, Ele_field_mtx + i*Ele_field_mtx_N + N3, N12, N3);
		mtx_prod(vec_temp + 2*N3, mtx_C.C_101_r, mtx_C.C_101_c, mtx_C.C_101_v, Ele_field_mtx + i*Ele_field_mtx_N + 2*N3, N12, N3);
		mtx_prod(vec_temp + 3*N3, mtx_C.C_011_r, mtx_C.C_011_c, mtx_C.C_011_v, Ele_field_mtx + i*Ele_field_mtx_N + 3*N3, N12, N3);
		mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_temp, N12, N3, "Conjugate Transpose");
		mtx_prod(vec_left + N3, mtx_C.C_110_r, mtx_C.C_110_c, mtx_C.C_110_v, vec_temp + N3, N12, N3, "Conjugate Transpose");
		mtx_prod(vec_left + 2*N3, mtx_C.C_101_r, mtx_C.C_101_c, mtx_C.C_101_v, vec_temp + 2*N3, N12, N3, "Conjugate Transpose");
		mtx_prod(vec_left + 3*N3, mtx_C.C_011_r, mtx_C.C_011_c, mtx_C.C_011_v, vec_temp + 3*N3, N12, N3, "Conjugate Transpose");
   
   		vec_plus(residual, 1.0, vec_left, omega2, Dis_field_mtx + i*Ele_field_mtx_N, Ele_field_mtx_N);
		res = vec_norm(residual, Ele_field_mtx_N);

		if(res > error)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				//Freq_array[i] = -Freq_array[i];
		}
		else
		{
			printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
		}

	}

	free(vec_temp); free(vec_left); free(residual);
	
}

void Check_Residual_Biiso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error)
{
	int mtx_N = N * 6;
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size;
	
	size = mtx_N * Nwant * sizeof(cmpxCPU);
	cmpxCPU scal=0.0+1.0*I;

	cmpxCPU* vec_temp = (cmpxCPU*)malloc(size);
	cmpxCPU* vec_left = (cmpxCPU*)malloc(size);
	cmpxCPU* residual = (cmpxCPU*)malloc(size);

	realCPU res, omega2;
	realCPU* B_eps = (realCPU*)calloc(N3, sizeof(realCPU));
	cmpxCPU* B_zeta = (cmpxCPU*)calloc(N3, sizeof(cmpxCPU));
	realCPU* B_mu = (realCPU*)calloc(N3, sizeof(realCPU));
	cmpxCPU* B_xi = (cmpxCPU*)calloc(N3, sizeof(cmpxCPU));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, N3*sizeof(realCPU), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_zeta, mtx_B.B_zeta, N3*sizeof(cmpxGPU), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_mu, mtx_B.B_mu, N3*sizeof(realCPU), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_xi, mtx_B.B_xi, N3*sizeof(cmpxGPU), hipMemcpyDeviceToHost));


	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -Freq_array[i];

		mtx_dot_prod(B_zeta, Ele_field_mtx + i*mtx_N, residual, N3, 1);
		mtx_dot_prod(B_mu, Ele_field_mtx + i*mtx_N + N3, vec_left, N3, 1);
		mtx_dot_prod(B_eps, Ele_field_mtx + i*mtx_N, residual + N3, N3, 1);
		mtx_dot_prod(B_xi, Ele_field_mtx + i*mtx_N + N3, vec_left + N3, N3, 1);


		vec_plus(vec_temp, scal, vec_left, scal, residual, N3);
		vec_plus(vec_temp + N3, -scal, vec_left + N3, -scal, residual + N3, N3);

		mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N, N12, N3);
		mtx_prod(vec_left + N3, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N + N3, N12, N3, "Conjugate Transpose");
			
		vec_plus(residual, 1.0, vec_left, omega2, vec_temp, mtx_N);

		res = vec_norm(residual, mtx_N);

		if(res > error)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				//Freq_array[i] = -Freq_array[i];
		}
		else
			printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
	}

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);free(B_zeta);free(B_xi);free(B_mu);
	
}

void Check_Residual_Bianiso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, cmpxCPU* Dis_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant, realCPU error)
{
	int mtx_N = N * 48;
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size;
	
	size = mtx_N * Nwant * sizeof(cmpxCPU);
	cmpxCPU scal = 0.0 + 1.0*I;

	cmpxCPU* vec_left = (cmpxCPU*)malloc(size);
	cmpxCPU* residual = (cmpxCPU*)malloc(size);

	realCPU res;
	cmpxCPU omega2;

	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -Freq_array[i] * scal;

		mtx_prod(vec_left + 0 * N3,  mtx_C.C_r,     mtx_C.C_c,     mtx_C.C_v,     Ele_field_mtx + i * mtx_N + 12 * N3, N12, N3, "minus", "Conjugate Transpose");
		mtx_prod(vec_left + 1 * N3,  mtx_C.C_110_r, mtx_C.C_110_c, mtx_C.C_110_v, Ele_field_mtx + i * mtx_N + 13 * N3, N12, N3, "minus", "Conjugate Transpose");
		mtx_prod(vec_left + 2 * N3,  mtx_C.C_101_r, mtx_C.C_101_c, mtx_C.C_101_v, Ele_field_mtx + i * mtx_N + 14 * N3, N12, N3, "minus", "Conjugate Transpose");
		mtx_prod(vec_left + 3 * N3,  mtx_C.C_011_r, mtx_C.C_011_c, mtx_C.C_011_v, Ele_field_mtx + i * mtx_N + 15 * N3, N12, N3, "minus", "Conjugate Transpose");

		mtx_prod(vec_left + 4 * N3,  mtx_C.C_r,     mtx_C.C_c,     mtx_C.C_v,     Ele_field_mtx + i * mtx_N + 8 * N3, N12, N3, "Conjugate Transpose");
		mtx_prod(vec_left + 5 * N3,  mtx_C.C_110_r, mtx_C.C_110_c, mtx_C.C_110_v, Ele_field_mtx + i * mtx_N + 9 * N3, N12, N3, "Conjugate Transpose");
		mtx_prod(vec_left + 6 * N3,  mtx_C.C_101_r, mtx_C.C_101_c, mtx_C.C_101_v, Ele_field_mtx + i * mtx_N + 10 * N3, N12, N3, "Conjugate Transpose");
		mtx_prod(vec_left + 7 * N3,  mtx_C.C_011_r, mtx_C.C_011_c, mtx_C.C_011_v, Ele_field_mtx + i * mtx_N + 11 * N3, N12, N3, "Conjugate Transpose");

		mtx_prod(vec_left + 8 * N3,  mtx_C.C_r,     mtx_C.C_c,     mtx_C.C_v,     Ele_field_mtx + i * mtx_N + 4 * N3, N12, N3, "minus");
		mtx_prod(vec_left + 9 * N3,  mtx_C.C_110_r, mtx_C.C_110_c, mtx_C.C_110_v, Ele_field_mtx + i * mtx_N + 5 * N3, N12, N3, "minus");
		mtx_prod(vec_left + 10 * N3, mtx_C.C_101_r, mtx_C.C_101_c, mtx_C.C_101_v, Ele_field_mtx + i * mtx_N + 6 * N3, N12, N3, "minus");
		mtx_prod(vec_left + 11 * N3, mtx_C.C_011_r, mtx_C.C_011_c, mtx_C.C_011_v, Ele_field_mtx + i * mtx_N + 7 * N3, N12, N3, "minus");

		mtx_prod(vec_left + 12 * N3, mtx_C.C_r,     mtx_C.C_c,     mtx_C.C_v,     Ele_field_mtx + i * mtx_N + 0 * N3, N12, N3);
		mtx_prod(vec_left + 13 * N3, mtx_C.C_110_r, mtx_C.C_110_c, mtx_C.C_110_v, Ele_field_mtx + i * mtx_N + 1 * N3, N12, N3);
		mtx_prod(vec_left + 14 * N3, mtx_C.C_101_r, mtx_C.C_101_c, mtx_C.C_101_v, Ele_field_mtx + i * mtx_N + 2 * N3, N12, N3);
		mtx_prod(vec_left + 15 * N3, mtx_C.C_011_r, mtx_C.C_011_c, mtx_C.C_011_v, Ele_field_mtx + i * mtx_N + 3 * N3, N12, N3);
			
		vec_plus(residual, 1.0, vec_left, omega2, Dis_field_mtx + i * mtx_N, mtx_N);
		res = vec_norm(residual, mtx_N);

		if(res > error)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				//Freq_array[i] = -Freq_array[i];
		}
		else
		{
			printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
		}

	}

	free(vec_left); free(residual);
	
}

