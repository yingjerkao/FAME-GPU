#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <complex.h>
#include <assert.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"

void printDeviceArray(realGPU *d_Array, int n, const char *filename)
{
	hipError_t cudaErr;

    FILE *fp = fopen(filename, "w");
    assert( fp != NULL );

    printf("Write array into %s.\n", filename);

    realCPU *h_Array = (realCPU*) malloc( n * sizeof(realCPU) );
    cudaErr = hipMemcpy(h_Array, d_Array, n*sizeof(realCPU), hipMemcpyDeviceToHost);
	assert( cudaErr == hipSuccess );

	for(int i = 0 ; i < n; i++)
        fprintf(fp, "%15.18lf\n", h_Array[i]);
        
    fclose(fp);
    free(h_Array);
}

void printDeviceArray(cmpxGPU *d_Array, int n, const char *filename)
{
	hipError_t cudaErr;
    FILE *fp = fopen(filename, "w");
    assert( fp != NULL );

    printf("Write array into %s.\n", filename);

    cmpxCPU *h_Array = (cmpxCPU*) malloc( n * sizeof(cmpxCPU) );
    
    cudaErr = hipMemcpy(h_Array, d_Array, n*sizeof(cmpxGPU), hipMemcpyDeviceToHost);
    //printf("cudaErr %s.\n" , hipGetErrorString(cudaErr));
	assert( cudaErr == hipSuccess );

    for(int i = 0; i < n; i++)
       fprintf(fp, "%15.18lf\t %15.18lf\n", creal(h_Array[i]), cimag(h_Array[i]) );

    fclose(fp);
    free(h_Array);
}

void printDeviceArray(int *d_Array, int n, const char *filename)
{
	hipError_t cudaErr;
    FILE *fp = fopen(filename, "w");
    assert( fp != NULL );

    printf("Write array into %s.\n", filename);

    int *h_Array = (int*) malloc( n * sizeof(int) );
    cudaErr = hipMemcpy(h_Array, d_Array, n*sizeof(int), hipMemcpyDeviceToHost);
	assert( cudaErr == hipSuccess );	

    for(int i = 0; i < n; i++)
        fprintf(fp, "%d\n", h_Array[i]);

    fclose(fp);
    free(h_Array);
}

